#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fmt/core.h>

#include <atomic>
#include <chrono>
#include <cmath>
#include <random>
#include <stdexcept>
#include <string>
#include <vector>
#include <chrono>
#include <thread>

#include "hip/hip_runtime.h"

//#define DEBUG
#define THREADS_PER_BLOCK 256

//==================================================================================
// CUDA helper functions
//==================================================================================

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
int iDivUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//==================================================================================
// fast GPU PRNG functions
//==================================================================================
// efficient random numbers on GPU
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-37-efficient-random-number-generation-and-application

__device__ __inline__ unsigned TausStep(unsigned int &z, int S1, int S2, int S3, unsigned int M) {
  unsigned b = (((z << S1) ^ z) >> S2);
  return z = (((z & M) << S3) ^ b);
}
__device__ __inline__ unsigned LCGStep(unsigned int &z, unsigned int A, unsigned int C) { return z = (A * z + C); }
__device__ __inline__ float HybridTaus(unsigned int &z1, unsigned int &z2, unsigned int &z3, unsigned int &z4) {
  // Combined period is lcm(p1,p2,p3,p4)~ 2^121
  return float(2.3283064365387e-10) * (TausStep(z1, 13, 19, 12, 4294967294UL) ^ TausStep(z2, 2, 25, 4, 4294967288UL) ^
                                       TausStep(z3, 3, 11, 17, 4294967280UL) ^ LCGStep(z4, 1664525, 1013904223UL));
}

__device__ __inline__ float HybridTausSimple(unsigned int &z1, unsigned int &z2) {
  // Combined period is lcm(p1,p2)~ 2^60
  return float(2.3283064365387e-10) * (TausStep(z1, 13, 19, 12, 4294967294UL) ^ TausStep(z2, 2, 25, 4, 4294967288UL));
}

__device__ __inline__ float HybridTausSimplest(unsigned int &z1) {
  // Combined period is lcm(p1,p2)~ 2^30
  return float(2.3283064365387e-10) * TausStep(z1, 13, 19, 12, 4294967294UL);
}

//-------------------------------------------------------------------------

__device__ __inline__ unsigned int pcg_hash(unsigned int input) {
  unsigned int state = input * 747796405u + 2891336453u;
  unsigned int word = ((state >> ((state >> 28u) + 4u)) ^ state) * 277803737u;
  return (word >> 22u) ^ word;
}

__device__ __inline__ unsigned int rand_pcg(unsigned int rng_state) {
  unsigned int state = rng_state;
  rng_state = rng_state * 747796405u + 2891336453u;
  unsigned int word = ((state >> ((state >> 28u) + 4u)) ^ state) * 277803737u;
  return (word >> 22u) ^ word;
}

__device__ __inline__ unsigned int xorshift(unsigned int y) {
  // Liao et al 2020 SAGC "A 23.8Tbps Random Number Generator on a Single GPU"
  // https://github.com/L4Xin/quadruples-xorshift/
  y = y ^ (y << 11);
  y = y ^ (y >> 7);
  return y ^ (y >> 12);
}

__device__ __inline__ unsigned int xorshiftGM(unsigned int rng_state) {
  // Xorshift algorithm from George Marsaglia's paper
  rng_state ^= (rng_state << 13);
  rng_state ^= (rng_state >> 17);
  rng_state ^= (rng_state << 5);
  return rng_state;
}

__device__ __inline__ unsigned int xorshf96(unsigned int x) {
  unsigned int y = 362436069, z = 521288629;  // period 2^96-1
  unsigned int t;
  x ^= x << 16;
  x ^= x >> 5;
  x ^= x << 1;

  t = x;
  x = y;
  y = z;
  z = t ^ x ^ y;

  return z;
}

__device__ void testRNG(int n) {
  unsigned int rstate[4];
  for (int i = 0; i < 4; i++) rstate[i] = i * 12371;

  for (int i = 0; i < n; i++) {
    printf("%f\t", HybridTaus(rstate[0], rstate[1], rstate[2], rstate[3]));
  }
}

//==================================================================================
// actual MC gpu kernel
//==================================================================================

__global__ void mc_simulations_gpu_kernel(
    float *returns, const unsigned int n_returns, float *totals, const unsigned long N, const unsigned int n_periods) {
  // https://cvw.cac.cornell.edu/gpu/memory_arch
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= N) return;
//  printf("Hello from thread %d!\n", tid);

  //  // don't use hiprand b/c global memory
  //  https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-37-efficient-random-number-generation-and-application
  unsigned int rstate[] = {rand_pcg(tid), 21701, 1297, 65537};

  // first warp loads returns to shmem
  __shared__ float bufferReturns[1129];
  if (threadIdx.x < warpSize) {
    for (unsigned int i = threadIdx.x; i < n_returns; i += warpSize) {
      bufferReturns[i] = returns[i] * float(0.01);
    }
  }
  //  // second warp loads totals to shmem
  //  __shared__ float bufferTotals[THREADS_PER_BLOCK];  // should be blockDim.x
  //  if (threadIdx.x >= warpSize && threadIdx.x < 2 * warpSize) {
  //    for (unsigned int i = threadIdx.x - warpSize; i < THREADS_PER_BLOCK; i += warpSize) {
  //      //  if (threadIdx.x < warpSize) {
  //      //    for (unsigned int i = threadIdx.x; i < THREADS_PER_BLOCK; i +=
  //      //    warpSize) {
  //      bufferTotals[i] = totals[blockIdx.x * blockDim.x + i];
  //    }
  //  }
  __syncthreads();
  float total = totals[tid];
  //  float total = bufferTotals[threadIdx.x];

  // todo Sobol PRNG to avoid shmem bank conflicts?
  // https://github.com/NVIDIA/cuda-samples/tree/2e41896e1b2c7e2699b7b7f6689c107900c233bb/Samples/5_Domain_Specific/SobolQRNG

  // https://www.reedbeta.com/blog/quick-and-easy-gpu-random-numbers-in-d3d11/
  unsigned int prng_state = rand_pcg(tid);

  unsigned int return_idx;
  for (unsigned int i = 0; i < n_periods; i++) {
    //    return_idx = n_returns * HybridTaus(rstate[0], rstate[1], rstate[2], rstate[3]);
    //        return_idx = n_returns_f * HybridTausSimple(rstate[0], rstate[1]);
    prng_state = xorshift(prng_state);
    return_idx = n_returns * (prng_state * powf(2, -32));
    total += total * bufferReturns[return_idx];
  }
  totals[tid] = total;
  //  bufferTotals[threadIdx.x] = total;
  //  __syncthreads();
  //
  //  // write results from this block to global memory, coalesced
  //  if (threadIdx.x < warpSize) {
  //    for (unsigned int i = threadIdx.x; i < blockDim.x; i += warpSize) {
  //      totals[blockIdx.x * blockDim.x + i] = bufferTotals[i];
  //    }
  //  }
}

struct Plan {
  unsigned long n;
  float *returns_d;
  float *totals_d;
};

void create_plan(Plan &plan, int gpu_id, unsigned long N, unsigned int n_returns) {
  hipSetDevice(gpu_id);
  plan.n = N;
  gpuErrchk(hipMalloc(&(plan.returns_d), n_returns * sizeof(float)));
  gpuErrchk(hipMalloc(&(plan.totals_d), N * sizeof(float)));
}


void _mc_simulations_multi_gpu(float *returns,
                               const unsigned int n_returns,
                               float *totals,
                               const unsigned long N,
                               const unsigned int n_periods,
                               unsigned int n_gpus) {
  // TODO see https://stackoverflow.com/questions/11673154/concurrency-in-cuda-multi-gpu-executions/35010019#35010019

  //-----------------------
  printf("Allocating memory...");
  unsigned long n_todo = N;
  Plan plan[n_gpus];
  for (int dev = 0; dev < n_gpus; dev++) {
    printf("\tgpu %d", dev);
    unsigned long n_this_gpu = std::min(n_todo, N / n_gpus);
    n_todo = N - n_this_gpu;
    // allocate memory on the correct GPU device
    printf("-> will run %ld simulations", n_this_gpu);
    create_plan(plan[dev], dev, n_this_gpu, n_returns);
  }
  printf("\n");
  //----------------------
  printf("Transferring data ...");
  unsigned long n_done = 0;
  for (int dev = 0; dev < n_gpus; dev++) {
    printf("\tgpu %d", dev);
    hipSetDevice(dev);
    hipMemcpy(plan[dev].returns_d, returns, n_returns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(plan[dev].totals_d, totals + n_done, plan[dev].n * sizeof(float), hipMemcpyHostToDevice);
    n_done += dev * plan[dev].n;
  }
  printf("\n");
  //----------------------
  printf("Launching kernels...\n");
  for (int dev = 0; dev < n_gpus; dev++) {
    hipSetDevice(dev);
    dim3 block, grid;
    int block_size = THREADS_PER_BLOCK;
    block.x = block_size;
    grid.x = (plan[dev].n + block_size - 1) / block_size;
    printf("\t GPU %d -> block_no: %d, block_size: %d | warps/block: %d \n", dev, grid.x, block.x, block.x / 32);
    mc_simulations_gpu_kernel<<<grid, block>>>(
        plan[dev].returns_d, n_returns, plan[dev].totals_d, plan[dev].n, n_periods);
  }
  printf("\n");
  hipDeviceSynchronize();
  //----------------------
  printf("Gathering results...");
  n_done = 0;
  for (int dev = 0; dev < n_gpus; dev++) {
    printf("\tgpu %d", dev);
    hipSetDevice(dev);
    hipMemcpy(totals + n_done, plan[dev].totals_d, plan[dev].n * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree(plan[dev].returns_d);
//    hipFree(plan[dev].totals_d);
    n_done += plan[dev].n;
  }
  printf("\n");
  hipDeviceReset();
}

//void _mc_simulations_multi_gpu(float *returns,
//                               const unsigned int n_returns,
//                               float *totals,
//                               const unsigned long N,
//                               const unsigned int n_periods,
//                               int n_gpus) {
//  // TODO see https://stackoverflow.com/questions/11673154/concurrency-in-cuda-multi-gpu-executions/35010019#35010019
//
//  int dev = 0;
//  //-----------------------
//  printf("Allocating memory...");
//  unsigned long n_todo = N;
//  Plan plan[n_gpus];
//  create_plan(plan[dev], dev, N, n_returns);
//  printf("\n");
//
//  //----------------------
//  printf("Transferring data ...");
//  unsigned long n_done = 0;
//  gpuErrchk(hipSetDevice(dev));
//  gpuErrchk(hipMemcpy(plan[dev].returns_d, returns, n_returns * sizeof(float), hipMemcpyHostToDevice));
//  gpuErrchk(hipMemcpy(plan[dev].totals_d, totals, plan[dev].n * sizeof(float), hipMemcpyHostToDevice));
//  n_done += plan[dev].n;
//  printf("\n");
//
//  //----------------------
//  printf("Launching kernels...\n");
//  gpuErrchk(hipSetDevice(dev));
//  dim3 block, grid;
//  int block_size = THREADS_PER_BLOCK;
//  block.x = block_size;
//  grid.x = (plan[dev].n + block_size - 1) / block_size;
//  printf("\t GPU %d -> block_no: %u, block_size: %d | warps/block: %d \n", dev, grid.x, block.x, block.x / 32);
//  mc_simulations_gpu_kernel<<<grid, block>>>(
//      plan[dev].returns_d, n_returns, plan[dev].totals_d, plan[dev].n, n_periods);
//  printf("\n");
//  gpuErrchk(hipDeviceSynchronize());
//
//  //----------------------
//  printf("Gathering results...");
//  n_done = 0;
//  printf("\tgpu %d", dev);
//  gpuErrchk(hipSetDevice(dev));
//  gpuErrchk(hipMemcpy(totals + n_done, plan[dev].totals_d, plan[dev].n * sizeof(float), hipMemcpyDeviceToHost));
////  hipFree(plan[dev].returns_d);
////  hipFree(plan[dev].totals_d);
////  n_done += plan[dev].n;
//  printf("Done!\n");
////  std::this_thread::sleep_for(std::chrono::milliseconds(5 * 1000));;
//  gpuErrchk(hipDeviceReset());
//}

void _mc_simulations_gpu(
    float *returns, const unsigned int n_returns, float *totals, const unsigned long N, const unsigned int n_periods) {
  int block_size = THREADS_PER_BLOCK;
  dim3 block, grid;
  block.x = block_size;
  grid.x = (N + block_size - 1) / block_size;

  printf("block_no: %d, block_size: %d | warps/block: %d \n", grid.x, block.x, block.x / 32);

  hipSetDevice(0);
  //-----------------------
  // Memory allocations
  //----------------------
  float *returns_d, *totals_d;
  long memsize_hist_returns = n_returns * sizeof(float);
  long memsize_totals = N * sizeof(float);


  hipMalloc(&returns_d, memsize_hist_returns);
  hipMalloc(&totals_d, memsize_totals);

  hipMemcpy(returns_d, returns, memsize_hist_returns, hipMemcpyHostToDevice);
  hipMemcpy(totals_d, totals, memsize_totals, hipMemcpyHostToDevice);

  // launch kernel!
  mc_simulations_gpu_kernel<<<grid, block>>>(returns_d, n_returns, totals_d, N, n_periods);

  hipDeviceSynchronize();
  hipMemcpy(totals, totals_d, memsize_totals, hipMemcpyDeviceToHost);
  hipFree(returns_d);
  hipFree(totals_d);
}

void mc_simulations_gpu(std::atomic<unsigned long> &n_simulations,
                        const unsigned long N,
                        const unsigned int n_periods,
                        const float initial_capital,
                        std::vector<float> &returns,
                        std::vector<float> &final_values) {
  // initialize output array
  std::fill(final_values.begin(), final_values.end(), initial_capital);
  // get pointers b/c GPU can't use std::vectors
  float *totals_arr = &final_values[0];
  float *returns_arr = &returns[0];

//    _mc_simulations_gpu(returns_arr, returns.size(), totals_arr, N, n_periods);
  _mc_simulations_multi_gpu(returns_arr, returns.size(), totals_arr, N, n_periods, 1);

  n_simulations = N;  // TODO increment inside GPU kernel?
  // assert(n_simulations = N); // must be true here
}
