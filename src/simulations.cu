#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <atomic>
#include <vector>
#include <chrono>
#include <fmt/core.h>

//#define DEBUG

__device__ void update_fund(float fund_value, float period_return, float &next_value) {
  next_value = fund_value * (float(100.0) + period_return) / 100;
}

__global__ void __many_updates_gpu_kernel(float *returns, float *totals, long n_periods) {

  for (int i = 0; i < n_periods; i++) {
    update_fund(totals[i], returns[i], totals[i + 1]);
  }
}

// host function that allocates memory and calls the GPU
void __many_updates_gpu(float *returns, float *totals, long n) {
#ifdef DEBUG
  printf("Allocating device memory on host..\n");
#endif
  float *returns_d, *totals_d;
  hipMalloc((void **) &returns_d, n * sizeof(float));
  hipMalloc((void **) &totals_d, (n + 1) * sizeof(float));

#ifdef DEBUG
  printf("Copying to device..\n");
#endif
  hipMemcpy(returns_d, returns, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(totals_d, totals, (n + 1) * sizeof(float), hipMemcpyHostToDevice);

  int block_size = 32; // should always be multiple of 32, max 1024
  int block_no = ceil(float(n) / block_size);
  dim3 grid(block_no, 1, 1);
  dim3 block(block_size, 1, 1); // max block dimensions: [1024,1024,64]

#ifdef DEBUG
  printf("block_no: %d, block_size: %d | \n", block_no, block_size);
  clock_t start_d = clock();
#endif
  __many_updates_gpu_kernel<<<grid, block>>>(returns_d, totals_d, n);
  hipDeviceSynchronize();

#ifdef DEBUG
  clock_t end_d = clock();
  double time_d = (double) (end_d - start_d) / CLOCKS_PER_SEC;
  printf("GPU time: %f\n", time_d);
#endif

  hipMemcpy(totals, totals_d, (n + 1) * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(returns_d);
  hipFree(totals_d);
}

////////////////////////////////
__global__ void mc_simulations_gpu_kernel(float *historical_returns,
                                          const long n_historical_returns,
                                          float *totals,
                                          const long max_n_simulations,
                                          const long n_periods) {
  // threads in a block are on the same SM
  long id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= max_n_simulations)
    return;

  // http://ianfinlayson.net/class/cpsc425/notes/cuda-random
  hiprandState_t state;
  hiprand_init(12345, /* the seed can be the same for each core, here we pass the time in from the CPU */
              id, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);

  // every step, sample random return and update total
  for (int i = 0; i < n_periods; i++) {
    long return_idx = long(n_historical_returns * hiprand_uniform(&state));
    update_fund(totals[id], historical_returns[return_idx], totals[id]);
//    printf("%f -> %f\n", this_return, totals[id+i+1]);
  }
}

void _mc_simulations_gpu(float *historical_returns,
                         const long n_historical_returns,
                         float *totals,
                         const long max_n_simulations,
                         const long n_periods) {

  int block_size = 1024;
  int n_blocks = std::ceil(max_n_simulations / float(block_size));
  dim3 grid(n_blocks, 1, 1);
  dim3 block(block_size, 1, 1); // max block dimensions: [1024,1024,64]
  printf("block_no: %d, block_size: %d | \n", n_blocks, block_size);
  // TODO store historical_returns and/or totals in shared_memory?

  //allocations
  float *historical_returns_d, *totals_d;
  long memsize_hist_returns = n_historical_returns * sizeof(float);
  long memsize_totals = max_n_simulations * sizeof(float);

  hipMalloc((void **) &historical_returns_d, memsize_hist_returns);
  hipMalloc((void **) &totals_d, memsize_totals);

  hipMemcpy(historical_returns_d, historical_returns, memsize_hist_returns, hipMemcpyHostToDevice);
  hipMemcpy(totals_d, totals, memsize_totals, hipMemcpyHostToDevice);

  mc_simulations_gpu_kernel<<<grid, block>>>(historical_returns_d,
                                             n_historical_returns,
                                             totals_d,
                                             max_n_simulations,
                                             n_periods);
  hipDeviceSynchronize();
  hipMemcpy(totals, totals_d, memsize_totals, hipMemcpyDeviceToHost);
  hipFree(historical_returns_d);
  hipFree(totals_d);
}

void mc_simulations_gpu(std::atomic<long> &n_simulations,
                        const long max_n_simulations,
                        const long n_periods,
                        const float initial_capital,
                        std::vector<float> &historical_returns,
                        std::vector<float> &final_values) {

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

  // initialize output data array
  // must create as vector b/c: float total[max_n * n_periods] creates StackOverflow (SegFault) b/c on stack. Vectors are on heap
  std::vector<float> totals(max_n_simulations, initial_capital);
  float *totals_arr = &totals[0];
  float *historical_returns_arr = &historical_returns[0];

  _mc_simulations_gpu(historical_returns_arr,
                      historical_returns.size(),
                      totals_arr,
                      max_n_simulations,
                      n_periods);

  // save to vectors for further processing TODO avoid copy?
  for (long i = 0; i < max_n_simulations; i++) {
    final_values[i] = totals[i];
  }
  n_simulations = max_n_simulations; // TODO Increment inside GPU kernel?

  //assert(n_simulations = max_n_simulations); // must be true here

  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  auto timediff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
  fmt::print("All {} simulation done in {} s!\n", n_simulations, timediff / 1000.0);
}
